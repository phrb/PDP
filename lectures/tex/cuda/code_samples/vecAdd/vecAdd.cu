#include <stdio.h>
#include <hip/hip_runtime.h>


void checkErrors(hipError_t err, const char *msg) {
    if (err != hipSuccess) {
        fprintf(stderr, msg);
        fprintf(stderr,
                " [Erro CUDA: %s]\n",
                hipGetErrorString(err));
        exit(-1);
    }
}

void compareResults(float *C1, float *C2, int numElements) {
    float epsilon = 0.00001;
    for(int i = 0; i < numElements; i ++) {
        if (abs(C1[i] - C2[i]) > epsilon) {
            printf("Comparação de resultados falhou\n");
            exit(-1);
        }
    }
    printf("Comparação de resultados passou\n");
}

void vecAddCPU(float *A, float *B, float *C, int numElements) {
    for(int i = 0; i < numElements; i ++) {
        C[i] = A[i] + B[i];
    }
}

__global__ void vecAdd(float *A, float *B, float *C, int numElements) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements) {
        C[i] = A[i] + B[i];
    }
}

int main(int argc, char **argv) {
    int maxBlockSize = 1024;
    int numElements = 50000;
    int size = numElements * sizeof(float);

    printf("Alocando vetores no host\n");

    float *h_A = (float *) malloc(size);
    float *h_B = (float *) malloc(size);
    float *h_C = (float *) malloc(size);

    if (h_A == NULL || h_B == NULL || h_C == NULL) {
        fprintf(stderr, "Falha em alocar vetores no host\n");
        exit(-1);
    }

    printf("Inicializando vetores no host\n");

    for (int i = 0; i < numElements; ++i) {
        h_A[i] = rand() / (float) (RAND_MAX);
        h_B[i] = rand() / (float) (RAND_MAX);
    }

    float *d_A;
    float *d_B;
    float *d_C;

    printf("Alocando vetores no device\n");

    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    checkErrors(hipGetLastError(),
                "Malloc nos vetores do device");

    printf("Copiando memória do host para o device\n");

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    checkErrors(hipGetLastError(),
                "Cópia para o device");

    /*
      Precisamos de um número inteiro de  blocks, mesmo se "numElements" não for
      divisível por "maxBlockSize"
    */
    int numBlocks = (numElements + maxBlockSize - 1) / maxBlockSize;

    printf("Lançando um kernel com %d threads, com %d blocks de tamanho %d\n",
           numBlocks * maxBlockSize,
           numBlocks,
           maxBlockSize);

    vecAdd<<<numBlocks, maxBlockSize>>>(d_A, d_B, d_C, numElements);

    checkErrors(hipGetLastError(),
                "Lançamento do kernel");

    printf("Copiando memória do device para o host\n");

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    checkErrors(hipGetLastError(),
                "Cópia para o host");

    printf("Alocando vetor de teste no host\n");

    float *h_D = (float *) malloc(size);

    if (h_D == NULL) {
        fprintf(stderr, "Falha em alocar vetores no host\n");
        exit(-1);
    }

    printf("Lançando cálculo na CPU\n");

    vecAddCPU(h_A, h_B, h_D, numElements);

    printf("Comparando resultados na CPU e na GPU\n");

    compareResults(h_C, h_D, numElements);

    printf("Liberando memória\n");

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    free(h_A);
    free(h_B);
    free(h_C);
    free(h_D);

    printf("Fim\n");

    return 0;
}
