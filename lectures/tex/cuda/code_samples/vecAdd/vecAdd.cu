#include <stdio.h>

#include <hip/hip_runtime.h>


__global__ void vecAdd(float *A, float *B, float *C, int numElements) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements) {
        C[i] = A[i] + B[i];
    }
}

int main(int argc, char **argv) {
    int numElements = 50000;
    int size = numElements * sizeof(float);

    float *h_A = (float *) malloc(size);
    float *h_B = (float *) malloc(size);
    float *h_C = (float *) malloc(size);

    if (h_A == NULL || h_B == NULL || h_C == NULL) {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(-1);
    }

    // Initialize the host input vectors
    for (int i = 0; i < numElements; ++i) {
        h_A[i] = rand() / (float) (RAND_MAX);
        h_B[i] = rand() / (float) (RAND_MAX);
    }

    float *d_A;
    float *d_B;
    float *d_C;

    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_B, size);

    // Copy the host input vectors A and B in host memory to the device input
    // vectors in device memory
    printf("Copy input data from the host memory to the CUDA device\n");
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;

    printf("CUDA kernel launch with %d blocks of %d threads\n",
           blocksPerGrid,
           threadsPerBlock);

    dim3 cudaBlockSize(threadsPerBlock, 1, 1);
    dim3 cudaGridSize(blocksPerGrid, 1, 1);

    hipError_t err = hipSuccess;
    vecAdd<<<cudaGridSize, cudaBlockSize>>>(d_A, d_B, d_C, numElements);
    err = hipGetLastError();

    if (err != hipSuccess) {
        fprintf(stderr,
                "Failed to launch vectorAdd kernel (error code %s)!\n",
                hipGetErrorString(err));
        exit(-1);
    }

    printf("Copy output data from the CUDA device to the host memory\n");
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    free(h_A);
    free(h_B);
    free(h_C);

    printf("Done!\n");

    return 0;
}
