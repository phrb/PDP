#include <stdio.h>
#include <hip/hip_runtime.h>


void checkErrors(hipError_t err, const char *msg) {
    if (err != hipSuccess) {
        fprintf(stderr, msg);
        fprintf(stderr,
                " [Erro CUDA: %s]\n",
                hipGetErrorString(err));
        exit(-1);
    }
}

void compareResults(float *C1, float *C2, int N) {
    float epsilon = 0.00001;
    for(int i = 0; i < N; i ++) {
        if (abs(C1[i] - C2[i]) > epsilon) {
            printf("Comparação de resultados falhou\n");
            exit(-1);
        }
    }
    printf("Comparação de resultados passou\n");
}

__global__ void matMul(float *A, float *B, float *C, int N) {
    int row = blockDim.x * blockIdx.x + threadIdx.x;
    int col = blockDim.y * blockIdx.y + threadIdx.y;

    if (row < N && col < N) {
        for(int i = 0; i < N; i++) {
            C[(row * N) + col] += A[(i * N) + col] * B[(row * N) + i];
        }
    }
}

void matMulCPU(float *A, float *B, float *C, int N) {
    for(int row = 0; row < N; row++) {
        for(int col = 0; col < N; col++) {
            for(int i = 0; i < N; i++) {
                C[(row * N) + col] += A[(i * N) + col] * B[(row * N) + i];
            }
        }
    }
}

int main(int argc, char **argv) {
    int maxBlockSize = 1024;
    int N = 256;
    int size = N * N * sizeof(float);

    int i, j;

    printf("Alocando vetores no host\n");

    float *h_A = (float *) malloc(size);
    float *h_B = (float *) malloc(size);
    float *h_C = (float *) malloc(size);

    if(h_A == NULL || h_B == NULL || h_C == NULL) {
        printf("Falha ao alocar memória\n");
    }

    for(i = 0; i < N; i++) {
        for(j = 0; j < N; j++) {
            h_A[(i * N) + j] = rand() / (float) (RAND_MAX);
            h_B[(i * N) + j] = rand() / (float) (RAND_MAX);
            h_C[(i * N) + j] = 0.0;
        }
    }

    float *d_A;
    float *d_B;
    float *d_C;

    printf("Alocando vetores no device\n");

    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    checkErrors(hipGetLastError(),
                "Malloc nos vetores do device");

    printf("Copiando memória do host para o device\n");

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    checkErrors(hipGetLastError(),
                "Cópia para o device");

    /*
      Precisamos de um número inteiro de  blocks, mesmo se "N" não for
      divisível por "maxBlockSize"
    */
    int blocksPerGrid = ((N * N) + maxBlockSize - 1) / maxBlockSize;

    dim3 cudaBlockSize(sqrt(maxBlockSize), sqrt(maxBlockSize), 1);
    dim3 cudaGridSize(sqrt(blocksPerGrid), sqrt(blocksPerGrid), 1);

    printf("Lançando um kernel com %d threads, com %d blocks de tamanho %d\n",
           blocksPerGrid * maxBlockSize,
           blocksPerGrid,
           maxBlockSize);

    matMul<<<cudaGridSize, cudaBlockSize>>>(d_A, d_B, d_C, N);

    checkErrors(hipGetLastError(),
                "Lançamento do kernel");

    printf("Copiando memória do device para o host\n");

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    checkErrors(hipGetLastError(),
                "Cópia para o host");

    printf("Alocando vetor de teste no host\n");

    float *h_D = (float *) malloc(size);

    if (h_D == NULL) {
        fprintf(stderr, "Falha em alocar vetores no host\n");
        exit(-1);
    }

    printf("Lançando cálculo na CPU\n");

    matMulCPU(h_A, h_B, h_D, N);

    printf("Comparando resultados na CPU e na GPU\n");

    compareResults(h_C, h_D, N);

    printf("Liberando memória\n");

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    free(h_A);
    free(h_B);
    free(h_C);
    free(h_D);

    printf("Fim\n");

    return 0;
}
